#include "MemoryBuffer.cuh"
#include <stdio.h>

EXTERN_C
{
	EXPORT int _HostToHostCopy(CMemoryBuffer dest, const CMemoryBuffer source)
	{
		return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyHostToHost);
	}

	EXPORT int _HostToDeviceCopy(CMemoryBuffer dest, const CMemoryBuffer source)
	{
		return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyHostToDevice);
	}

	EXPORT int _DeviceToHostCopy(CMemoryBuffer dest, const CMemoryBuffer source)
	{
		return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyDeviceToHost);
	}

	EXPORT int _DeviceToDeviceCopy(CMemoryBuffer dest, const CMemoryBuffer source)
	{
		return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyDeviceToDevice);
	}

	EXPORT int _AutoCopy(CMemoryBuffer dest, const CMemoryBuffer source)
	{
		return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyDefault);
	}

	EXPORT int _Alloc(CMemoryBuffer& buf)
	{
		int ret = hipMalloc((void **)&buf.pointer, buf.TotalSize());
		return ret;
	}

	EXPORT int _AllocHost(CMemoryBuffer& buf)
	{
		int ret = hipHostMalloc((void **)&buf.pointer, buf.TotalSize());
		return ret;
	}

	EXPORT int _Free(const CMemoryBuffer buf)
	{
		hipDeviceSynchronize();
		return hipFree((void *)buf.pointer);
	}

	EXPORT int _FreeHost(const CMemoryBuffer buf)
	{
		return hipHostFree((void *)buf.pointer);
	}
}